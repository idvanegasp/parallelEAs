// compilacion: nvcc 3_p6_collvile.cu -o collville

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <pthread.h>

#define POP_SIZE 2048
#define IND_SIZE 64
#define STEP 1024 
// bits = log2 STEP 
//16 - bits = numBits domain size representation
#define BITS 10
#define ITERATIONS 40000

double Hist_res[ITERATIONS];

struct Individuo
{
	char genes[IND_SIZE];
};


struct Individuo pop[POP_SIZE]; 

char* poblacion;
int fittest = 0; 
int secondFittest = 0; 
double aux_fittest[POP_SIZE];
int mutate_var = 0; // no booleans on C :0
double res = 0.0;


void generatePopulation();
void evaluate();
void select();
void cruce(); 
void mutate(); 
void imprimir_poblacion();
void imprimir_individuo(int idx);
void decodificar_individuo(int idx);

__global__ void evaluateKernel( char* pop, double* aux_fittest_gpu, int chrom_size, int pop_size, int step, int bits ){

	int h = threadIdx.x + blockDim.x*blockIdx.x ;
	///int h2 = threadIdx.y+ blockDim.y*blockIdx.y ;
	double fit_val = 0.0;
	double aux_X = 0.0;
	double aux_Y = 0.0;
	double aux_W = 0.0;
	double aux_Z = 0.0;

if (h < pop_size)
{
	for (int k = 0; k<chrom_size/4; k++)
		aux_X = aux_X+ pop[h]*powf(2 , ((chrom_size/4)-k)-1);
	aux_X < (aux_X/step) - powf(2,(chrom_size/4)-bits)/2;

	for (int k = chrom_size/4; k<chrom_size/2; k++)
		aux_Y = aux_Y+ pop[h]*powf(2 , ((chrom_size/2)-k)-1);
		aux_Y = (aux_Y/step) - powf(2,(chrom_size/4)-bits)/2;
	for (int k = chrom_size/2; k<chrom_size - 16; k++)
		aux_W = aux_W +pop[h]*powf(2 , ((chrom_size-16)-k)-1);
		aux_W = (aux_W/step) - powf(2,(chrom_size/4)-bits)/2;
	for (int k = (chrom_size-16); k<chrom_size; k++)
		aux_Z = aux_Z+ pop[h]*powf(2 , (chrom_size-k)-1);
		aux_Z = (aux_Z/step) - powf(2,(chrom_size/4)-bits)/2;

		fit_val = 100*powf((aux_Y - powf(aux_X,2)), 2) + powf((1-aux_X),2) + 90*powf((aux_Z-powf(aux_W,2)),2) + powf((1 - aux_W),2) +10.1*(powf((aux_Y-1),2) + powf((aux_Z-1),2));

		aux_fittest_gpu[h] = fit_val; 
}
	
}



//https://towardsdatascience.com/introduction-to-genetic-algorithms-including-example-code-e396e98d8bf3

// Inicio bloque principal

int main(void )
{

time_t t;
srand((unsigned int) time(&t));
hipDeviceProp_t deviceProp;
hipGetDeviceProperties(&deviceProp, 0);
generatePopulation();
printf("Población Inicial:\n");
//imprimir_poblacion();
printf("starting evolution ... %s" , deviceProp.name);

	hipSetDevice(0);
	// allocate device memory
	// allocat device memory
	size_t nBytes = sizeof(char *)*POP_SIZE*IND_SIZE;
	char* gpuRef; 
	gpuRef = (char*)malloc(nBytes);
	char* d_P;
	double* aux_fittest_gpu;// = (double *)malloc(POP_SIZE);
	poblacion = (char*)malloc(nBytes);
	memset(gpuRef,0, nBytes);
//	memset(aux_fittest_gpu, 0, POP_SIZE);
	hipMalloc((char** ) &d_P, nBytes);
	hipMalloc((double**) &aux_fittest_gpu, POP_SIZE);
	

	dim3 blockDim(32,32);
	dim3 gridDim(POP_SIZE/32,IND_SIZE/32);


for (int i = 0; i<ITERATIONS; i++)
{
hipMemcpy(d_P, poblacion, nBytes, hipMemcpyHostToDevice);
evaluateKernel<<<gridDim, blockDim>>>(d_P, aux_fittest_gpu, IND_SIZE, POP_SIZE, STEP, BITS);
hipDeviceSynchronize();
hipMemcpy(gpuRef, d_P, nBytes, hipMemcpyDeviceToHost);
hipMemcpy(aux_fittest, aux_fittest_gpu, nBytes, hipMemcpyDeviceToHost);
select();
Hist_res[i] = res;
cruce();
//imprimir_poblacion();
if (mutate_var)	mutate();
		
}
evaluate();
select();
//imprime evolucion del fitness de cada generacion
/*for (int i = 0; i<ITERATIONS; i++)
	printf("\nit %d\t%f", i, Hist_res[i]);
*/
hipFree(d_P);
hipFree(aux_fittest_gpu);

return 0;
}

// Fin bloque principal

void generatePopulation(){

	for (int i = 0; i<POP_SIZE; i++)
	{
		strcpy( pop[i].genes, "\0");
	}
	int s; // random bit
	for (int j = 0; j < POP_SIZE; j++)
	{	
		for (int i = 0; i < IND_SIZE; i++)
		{
			s = rand()%2;
			pop[j].genes[i] = (char)s;
		}
	}

}


//res+= (p[i]*p[i]) - (10 * cos(p[i])) + 10; rastreging
// f(x,y) = x^2 = 2y^2 - 0.3cos(3pix) - 0.4cos(4piy) + 0.7
// f(x,y,w,z) = 100(y - x^2)^2 + (1-x)^2 +90(z-w^2)^2 + (1 - w)^2 +10.1((y-1)^2 + (z-1)^2)

void evaluate()
{

for (int j = 0; j < POP_SIZE; j++)
	{	
		double fit_val = 0.0;
		double aux_X = 0.0;
		double aux_Y = 0.0;
		double aux_W = 0.0;
		double aux_Z = 0.0;
		// bloques altamente paralelizables
		for (int i = 0; i < IND_SIZE/4; i++)
		{
			aux_X += pop[j].genes[i]*pow(2 , ((IND_SIZE/4)-i)-1); 

		}
		aux_X = (aux_X/STEP) - pow(2,(IND_SIZE/4)-BITS)/2;
		for (int i = IND_SIZE/4; i < IND_SIZE/2; i++)
		{
			aux_Y += pop[j].genes[i]*pow(2 , ((IND_SIZE/2)-i)-1); 

		}
		aux_Y = (aux_Y/STEP) - pow(2,(IND_SIZE/4)-BITS)/2;
		for (int i = IND_SIZE/2; i < IND_SIZE-16; i++)
		{
			aux_W += pop[j].genes[i]*pow(2 , (IND_SIZE-16-i)-1); 

		}
		aux_W = (aux_W/STEP) - pow(2,(IND_SIZE/4)-BITS)/2;
		for (int i = IND_SIZE-16 ; i < IND_SIZE; i++)
		{
			aux_Z += pop[j].genes[i]*pow(2 , (IND_SIZE-i)-1); 

		}
		aux_Z = (aux_Z/STEP) - pow(2,(IND_SIZE/4)-BITS)/2;
		
		//res += (pow(aux_X, 2)) - (10 * cos(2*aux_X)) + 10;
		//res += (pow(aux_Y, 2)) - (10 * cos(2*aux_Y)) + 10;
		
//		res = pow(x, 2) + 2*pow(y, 2) -0.3*cos(3*pi*x) -0.4*cos(4*pi*y) + 0.7;
		fit_val = 100*pow((aux_Y - pow(aux_X,2)), 2) + pow((1-aux_X),2) + 90*pow((aux_Z-pow(aux_W,2)),2) + pow((1 - aux_W),2) +10.1*(pow((aux_Y-1),2) + pow((aux_Z-1),2));
		//printf("X%d : %f, Y%d : %f;  W%d : %f;  Z%d : %f; f%d = %f\n", j+1,aux_X, j+1, aux_Y, j+1, aux_W, j+1, aux_Z, j+1, res);
		aux_fittest[j] = fit_val; 
	}


	
}


void select()
{
	res = 0.0;
	int aux_fit_1 = 0;
	for (int j = 1; j < POP_SIZE; j++)
	{
		if (aux_fittest[j] < aux_fittest[aux_fit_1]){
			aux_fit_1 = j;		
		}
	}
	fittest = aux_fit_1;
	int aux_fit_2 = 0;
	if (aux_fit_1 == 0) aux_fit_2++;
	for (int j = aux_fit_2+1; j < POP_SIZE; j++)
	{
		if (j != aux_fit_1 && aux_fittest[j] < aux_fittest[aux_fit_2]){
			aux_fit_2 = j;		
		}
	}	
	secondFittest = aux_fit_2;
	if(abs(aux_fittest[aux_fit_1]-aux_fittest[aux_fit_2]) < 0.00001) mutate_var = 1;
	//printf("El individuo minimo es %d y el segundo %d\n", fittest+1, secondFittest+1);
	res=aux_fittest[fittest];
	decodificar_individuo(fittest);
}
void cruce()
{

	int s;
	s = rand()%IND_SIZE;
	
	char aux_ind_1[IND_SIZE];
	char aux_ind_2[IND_SIZE];

//	strcpy( aux_ind_1, &pop[fittest].genes);
//	strcpy( aux_ind_2, &pop[secondFittest].genes);
	
	for (int i = 0; i < IND_SIZE; i++)
	{
		aux_ind_1[i] = pop[fittest].genes[i];
		aux_ind_2[i] = pop[secondFittest].genes[i];
	}
	printf("\n");

	// crea los dos nuevos individuos del offspring y actualiza la poblacion
	struct Individuo new_pop[POP_SIZE]; 
	for (int i = 0; i<POP_SIZE; i++)
	{
		strcpy( new_pop[i].genes, "\0");
	}

// crea nueva poblacion
	for (int j = 0; j<POP_SIZE; j+=2)
	{
		for (int i = 0; i < IND_SIZE; i++)
		{
			new_pop[j].genes[i] = aux_ind_1[i];
			new_pop[j+1].genes[i] = aux_ind_2[i];
		}
	}
// actualiza la info de los hijos
	for (int j = 2; j<POP_SIZE; j+=2)
	{
		s = rand()%IND_SIZE;
		for (int i = s+1; i < IND_SIZE; i++)
		{
			new_pop[j].genes[i] = aux_ind_2[i];
			new_pop[j+1].genes[i] = aux_ind_1[i];
		}
	}
	memcpy(&pop, &new_pop, sizeof new_pop);
}


void mutate()
{
	for ( int i = 1; i < POP_SIZE; i++)
	{

	//numvars
	for ( int j = 0 ; j< IND_SIZE/16 ; j++)

	{
		int mut = rand()%2;
		if (mut)
		{
			int k = rand()%IND_SIZE/4 ;
			if(pop[i].genes[(j*IND_SIZE/4) + k]%2==1)
			{
				pop[i].genes[(j*IND_SIZE/4) + k] = 0;
	//			printf("Muta Ind%d cromosoma%d\n", i+1, k+1);
			}else pop[i].genes[(j*IND_SIZE/4) + k] = 1;
		}
	
	}

		// exchange rows
	}
	mutate_var = 0;
}

void imprimir_individuo(int k)
{
	for (int i = 0; i < IND_SIZE; i++)
	{
		printf("%d", pop[k].genes[i]);
	}
	printf("\n");
}
void imprimir_poblacion()
{
	for (int j = 0; j < POP_SIZE; j++)
	{
		for (int i = 0; i < IND_SIZE; i++)
		{
			printf("%d", pop[j].genes[i]);
		}
		printf("\n");
	}
	printf("\n");
	printf("\n");
}

void decodificar_individuo(int k)
{
	imprimir_individuo(k);
	double aux_X = 0.0;
	double aux_Y = 0.0;
	double aux_W = 0.0;
	double aux_Z = 0.0;

/// mejorar: X_Y_W_Z[] = funcion_decodificarIndividuo(chrom, res[])
	for (int i = 0; i < IND_SIZE/4; i++)
	{
		aux_X += pop[k].genes[i]*pow(2 , ((IND_SIZE/4)-i)-1); 
	}
	aux_X = (aux_X/STEP) - pow(2,(IND_SIZE/4)-BITS)/2;
	for (int i = IND_SIZE/4; i < IND_SIZE/2; i++)
	{
		aux_Y += pop[k].genes[i]*pow(2 , ((IND_SIZE/2)-i)-1); 
	}
	aux_Y = (aux_Y/STEP) - pow(2,(IND_SIZE/4)-BITS)/2;
	for (int i = IND_SIZE/2; i < IND_SIZE-16; i++)
	{
		aux_W += pop[k].genes[i]*pow(2 , (IND_SIZE-16-i)-1); 
		}
	aux_W = (aux_W/STEP) - pow(2,(IND_SIZE/4)-BITS)/2;
	for (int i = IND_SIZE-16 ; i < IND_SIZE; i++)
	{
		aux_Z += pop[k].genes[i]*pow(2 , (IND_SIZE-i)-1); 
	}
	aux_Z = (aux_Z/STEP) - pow(2,(IND_SIZE/4)-BITS)/2;
	printf("X: %f Y: %f W: %f Z: %f\n fitness %f\n", aux_X, aux_Y, aux_W, aux_Z, res);
}
